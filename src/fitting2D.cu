#include "hip/hip_runtime.h"
#include "fitting2D.hpp"

__global__ 
void lineFittingCUDA(float* pointsArr, int* inlinerCounts, int pointsNum, int maxIterations, float delta) {
    // warps are models, so threads search the points divided by 32
    int warpIdx = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int laneIdx = (blockIdx.x * blockDim.x + threadIdx.x) % WARP_SIZE;

    // TODO: use shared memory

    // model generation
    // the first point is the leader thread's warpIdx % pointsNum
    // the second point is the leader thread's warpIdx % pointsNum + warpIdx / pointsNum
    int firstPointIdx = warpIdx % pointsNum;
    int secondPointIdx = warpIdx % pointsNum + warpIdx / pointsNum + 1;
    if (secondPointIdx >= pointsNum) {
        secondPointIdx -= pointsNum;
    }

    float Ox = pointsArr[firstPointIdx * 2];
    float Oy = pointsArr[firstPointIdx * 2 + 1];
    float Rx = pointsArr[secondPointIdx * 2] - Ox;
    float Ry = pointsArr[secondPointIdx * 2 + 1] - Oy;

    // each thread calculates the number of inliners and accumulates them at the end
    int offset = pointsNum / WARP_SIZE;
    int counter = 0;
#pragma unroll
    for (int i = 0; i < offset; i++) {
        int pointIdx = laneIdx * offset + i;
        float Qx = pointsArr[pointIdx * 2];
        float Qy = pointsArr[pointIdx * 2 + 1];
        float distance = abs(Rx * (Qy - Oy) - Ry * (Qx - Ox)) / sqrt(Rx * Rx + Ry * Ry);
        if (distance < delta) {
            counter++;
        }
    }

    // accumulate the inliner counts within a warp
    for (int i = 16; i > 0; i /= 2)
        counter += __shfl_down_sync(0xffffffff, counter, i);
    
    // write the inliner counts to the global memory
    if (laneIdx == 0) {
        inlinerCounts[warpIdx] = counter;
    }
}

__global__ 
void circleFittingCUDA(float* pointsArr, int* inlinerCounts, int pointsNum, int maxIterations, float delta) {
    ;
}


void Fitter2D::runFittingWithCUDA(PointCloudPtr& cloudCopy) {
    // point to array
    float* pointsArr = new float[cloudCopy->points.size() * 2];
    for (int i = 0; i < cloudCopy->points.size(); i++) {
        pointsArr[2*i] = (float)cloudCopy->points[i].x;
        pointsArr[2*i + 1] = (float)cloudCopy->points[i].y;
    }

    // shuffle the pointsArr with std::suffle
    std::random_device rd;
    std::mt19937 g(rd());
    for (auto i=cloudCopy->points.size()-1; i>0; --i) {
        std::uniform_int_distribution<decltype(i)> d(0,i);
        std::swap (pointsArr[i], pointsArr[d(g)]);
        std::swap (pointsArr[i+1], pointsArr[d(g)+1]);
    }

    // upload the pointsArr to GPUs
    float* pointsArr_d;
    hipMalloc((void**)&pointsArr_d, cloudCopy->points.size() * 2 * sizeof(float));
    hipMemcpy(pointsArr_d, pointsArr, cloudCopy->points.size() * 2 * sizeof(float), hipMemcpyHostToDevice);

    // setup inliner counts
    int* inlinerCounts = new int[m_maxIterations];
    int* inlinerCounts_d;
    hipMalloc((void**)&inlinerCounts_d, m_maxIterations * sizeof(int));
    hipMemset(inlinerCounts_d, 0, m_maxIterations * sizeof(int));

    // warp numbers are model numbers
    // thread block size is 256, fixed = 8 warps
    if (m_application == "line") {
        lineFittingCUDA<<<m_maxIterations / WARP_PER_BLOCK, BLOCK_SIZE>>>(pointsArr_d, inlinerCounts_d, cloudCopy->points.size(), m_maxIterations, m_delta);
    }
    else if (m_application == "circle") {
        circleFittingCUDA<<<m_maxIterations / WARP_PER_BLOCK, BLOCK_SIZE>>>(pointsArr_d, inlinerCounts_d, cloudCopy->points.size(), m_maxIterations, m_delta);
    }

    // download the inliner counts
    hipMemcpy(inlinerCounts, inlinerCounts_d, m_maxIterations * sizeof(int), hipMemcpyDeviceToHost);

    // get the best model
    int bestModelIdx = 0;
    int bestModelInlinerCount = 0;
    for (int i = 0; i < m_maxIterations; i++) {
        if (inlinerCounts[i] > bestModelInlinerCount) {
            bestModelIdx = i;
            bestModelInlinerCount = inlinerCounts[i];
        }

        // probability of success
        float p = m_threshold;
        float w = (float)inlinerCounts[i] / (float)m_maxIterations;
        float N = log(1 - p) / log(1 - pow(w, m_numRequiredPoints));
        if (N < m_maxIterations) {
            m_maxIterations = (int) N;
        }
    }

    // print the coordinates of the best model
    if (m_application == "line") {
        int firstPointIdx = bestModelIdx % cloudCopy->points.size();
        int secondPointIdx = firstPointIdx + bestModelIdx / cloudCopy->points.size() + 1;
        if (secondPointIdx >= cloudCopy->points.size()) {
            secondPointIdx -= cloudCopy->points.size();
        }
        float Ox = pointsArr[firstPointIdx * 2];
        float Oy = pointsArr[firstPointIdx * 2 + 1];
        float Rx = pointsArr[secondPointIdx * 2] - Ox;
        float Ry = pointsArr[secondPointIdx * 2 + 1] - Oy;

        Eigen::VectorXf model_(6);
        model_ << Ox, Oy, 0, Rx, Ry, 0;
        getBestModelCoefficients(model_);
    }
    else if (m_application == "circle") {
        ;
    }    

    // free the memory
    delete[] pointsArr;
    delete[] inlinerCounts;
    hipFree(pointsArr_d);
    hipFree(inlinerCounts_d);
}